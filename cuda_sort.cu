#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "./serial_sort.h"

extern "C" {
	void CU_Init();
	elem* CU_cudaAlloc(size_t bytes);
	void CU_cudaFree(elem* ptr);
	void CU_OddEvenNetworkSort(elem* begin, elem* end);
}

hipError_t cE;

void CU_Init(int world_rank, int world_size) {
	
	int assignedCudaDevice = -1;
	cE = hipSuccess;
	int cudaDeviceCount = -1;

	if ((cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess) {
		fprintf(stderr, " Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount);
		exit(EXIT_FAILURE);
	}
	
	if ((cE = hipSetDevice(world_rank % cudaDeviceCount)) != hipSuccess) {
		fprintf(stderr, " Unable to have rank %d set to cuda device %d, error is %d\n", world_rank, (world_rank % cudaDeviceCount), cE);
		exit(EXIT_FAILURE);
	}

	if ((cE = hipGetDevice(&assignedCudaDevice)) != hipSuccess) {
		fprintf(stderr, " Unable to have rank %d set to cuda device %d, error is %d\n", world_rank, (world_rank % cudaDeviceCount), cE);
		exit(EXIT_FAILURE);
	}

	if (assignedCudaDevice != (world_rank % cudaDeviceCount)) {
		fprintf(stderr, "MPI Rank %d: assignedCudaDevice: %d NOT EQUAL to (world_rank(%d) mod cudaDeviceCount(%d))\n", world_rank, assignedCudaDevice, world_rank, cudaDeviceCount);
		exit(EXIT_FAILURE);
	}
}

elem* CU_cudaAlloc(size_t bytes) {
	elem* ret = NULL;
	if ((cE = hipMallocManaged(&ret, bytes)) != hipSuccess) {
		fprintf(stderr, "ERROR: hipMallocManaged() failed with error code %d\n", cE);
		exit(EXIT_FAILURE);
	}
	return ret;
}	

void CU_cudaFree(elem* ptr) {
	if ((cE = hipFree(ptr)) != hipSuccess) {
		fprintf(stderr, "ERROR: hipFree() failed with error code %d\n", cE);
		exit(EXIT_FAILURE);
	}
}

__global__
void CU_OddEvenKernel(elem* arr, size_t n, int even, size_t numActions) {
	elem* a1 = NULL;
	elem* a2 = NULL;
	elem swp;

	size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	size_t incr = blockDim.x * gridDim.x;
	if (index >= numActions) {
		return;
	}
		
	for (; index < numActions; index += incr) {
		if (even) {
			a1 = arr + index * 2;
			a2 = arr + index * 2 + 1;
		} else {
			a1 = arr + index * 2 + 1;
			a2 = arr + index * 2 + 2;
		}

		if (((size_t)(a2 - arr)) < n && (*a2) < (*a1)) {
			swp = (*a1);
			(*a1) = (*a2);
			(*a2) = swp;
		}
	}
}

void CU_OddEvenNetworkSort(elem* begin, elem* end, size_t threadsCount) {

	size_t n = end - begin + 1;
	elem* cbegin;
	cE = hipMallocManaged(&cbegin, n * sizeof(elem));
	if (cE != hipSuccess) {
		fprintf(stderr, "ERROR: hipMallocManaged() failed with error code: %d\n", cE);
		exit(EXIT_FAILURE);
	}

	cE = hipMemcpy(cbegin, begin, n * sizeof(elem), ::hipMemcpyHostToDevice); 
	if (cE != hipSuccess) {
		fprintf(stderr, "ERROR: hipMemcpy(::HostToDevice) failed with error code: %d\n", cE);
		exit(EXIT_FAILURE);
	}

	ssize_t iters = n;
	while(iters--) {
		CU_OddEvenKernel<<<(n/2 + threadsCount)/threadsCount, threadsCount>>>(begin,
				n, (iters % 2) == 0, n/2);
		hipDeviceSynchronize();
	}

	cE = hipMemcpy(begin, cbegin, n * sizeof(elem), ::hipMemcpyDeviceToHost);
	if (cE != hipSuccess) {
		fprintf(stderr, "ERROR: hipMemcpy(::DeviceToHost) failed with error code: %d\n", cE);
		exit(EXIT_FAILURE);
	}

	cE = hipFree(cbegin);
	if (cE != hipSuccess) {
		fprintf(stderr, "ERROR: hipFree(::DeviceToHost) failed with error code: %d\n", cE);
		exit(EXIT_FAILURE);
	}
	
}

#if 0
int main(int argc, char** argv) {

	elem* arr = CU_cudaAlloc(5 * sizeof(elem));
	arr[0] = 5;
	arr[1] = 3;
	arr[2] = 4;
	arr[3] = 7;
	arr[4] = 1;
	
	printf("NUMS:");
	for (int i = 0; i < 5; ++i) {
		printf(" %d", arr[i]);
	}
	printf("\n");

	CU_OddEvenNetworkSort(arr, arr + 4, 3);

	printf("SORTED:");
	for (int i = 0; i < 5; ++i) {
		printf(" %d", arr[i]);
	}
	printf("\n");

	CU_cudaFree(arr);
	
}
#endif






















